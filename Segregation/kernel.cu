#include "hip/hip_runtime.h"
/* This file is part of SchellingSegregation: Exercise on the conventional Schelling segregation model.
	
SchellingSegregation is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

SchellingSegregation is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with SchellingSegregation.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace sim {
constexpr int nSigmaValues{100};

/// Represents the status of a cell.
enum class cellStatusEnum : char {
	empty,
	color1,	// Inhabited by a color1 person.
	color2	// Inhabited by a color2 person.
};

/// Represents a cell in space.
struct __align__(4) cellCls {
	cellStatusEnum status;
	char nAffineNeighbors;
};

/// Represents variables and components to the simulation that reside in shared memory.
struct worldCls {
	float sigma;
//	thrust::device_vector<cellCls> cells;
};

__global__ void kernel(float nNeighborsSigma[]) {
	__shared__ worldCls world;
	world.sigma = blockIdx.x / static_cast<float>(nSigmaValues);
	if(!threadIdx.x)
		nNeighborsSigma[blockIdx.x] = sizeof(size_t);
}

void main() {
	thrust::device_vector<float> nNeighborsSigma;
	nNeighborsSigma.resize(nSigmaValues, 0);
	kernel<<<nSigmaValues, 32>>>(thrust::raw_pointer_cast(nNeighborsSigma.data()));
	const thrust::host_vector<float> nNeighborsSigmaDevice{nNeighborsSigma};
	int sigmaI{0};
	for(const float nNeighSigma : nNeighborsSigmaDevice)
		std::cout << sigmaI++ / static_cast<float>(nSigmaValues) << "\t" << nNeighSigma << '\n';
}
} // namespace sim

int main() {
		sim::main();
		// hipDeviceReset must be called after every hipFree, such as after the destruction
		// of all the thrust::device_vector objects, and right before exiting, for profiling
		// and tracing tools such as Nsight and Visual Profiler to show complete traces.
		if(hipDeviceReset() != hipSuccess) {
			std::cerr << "hipDeviceReset failed. Please debug.\n";
			return 1;
		}
		return 0;
}
